
#include <hip/hip_runtime.h>
#include <stdio.h>

// The __global__ keyword is used to define
// functions (kernels) executed on the GPU,
// which can be called from the CPU. 
__global__ void hello_world()
{
    printf("Hello, World!\n");
}

int main()
{
    // Call the kernel. The <<<1, 1>>> syntax
    // indicates the # of blocks and # of threads:
    // <<< n_blocks, n_threads >>>
    hello_world<<<1, 1>>>();

    // Wait for all GPU processes to complete before
    // continuing program (analogous to MPI_Barrier)
    hipDeviceSynchronize();

    return 0;
}

// Note: To compile this CUDA code, type 
//
//    nvcc hello_world.cu -o main
//
// This will generate the following files:
// 1) main.exe
// 2) main.exp
// 3) main.lib
// The exp and lib files are only needed during
// compilation and can be deleted. To run the code,
// type ./main
